#include "hip/hip_runtime.h"
#define BOOST_TEST_MODULE HOW_CUFFT_WORKS
#include "boost/test/unit_test.hpp"

#ifndef FC_TRACE
#define FC_TRACE false
#endif

#include <numeric>
#include <vector>

#include "test_utils.hpp"
#include "image_stack_utils.h"
#include "traits.hpp"
#include "book.h"
#include "hipfft/hipfft.h"

namespace fourierconvolution {

  
  __global__ void scale(hipfftComplex* _array, size_t _size, float _scale){

    size_t tid = blockDim.x * blockIdx.x + threadIdx.x;

    hipfftComplex el;
    if(tid<_size){
      el = _array[tid];
      _array[tid].x = el.x*_scale;
      _array[tid].y = el.y*_scale;
    }
      

  }

  void inplace_fft_ifft(image_stack& _stack){

    
    typedef boost::multi_array<hipfftComplex,3> frequ_stack;
    
    const size_t img_size = _stack.num_elements();
    std::vector<size_t> shape(_stack.shape(),_stack.shape() + image_stack::dimensionality);
    
    BOOST_REQUIRE(img_size > 32);
  
    std::vector<size_t> shape_for_cufft(shape);
    shape_for_cufft[row_major::x] = (shape[row_major::x]/2) + 1;
    const size_t size_for_cufft = std::accumulate(shape_for_cufft.begin(), shape_for_cufft.end(),1,std::multiplies<size_t>());
  
    hipfftComplex* d_stack = 0;
  
    HANDLE_ERROR( hipMalloc( (void**)&(d_stack), size_for_cufft*sizeof(hipfftComplex) ) );
    HANDLE_ERROR( hipMemset( d_stack, 0, size_for_cufft*sizeof(hipfftComplex) ));

    //transform input data to cufft/fftw
    frequ_stack cufft_compliant(shape_for_cufft);
    float* stack_begin = _stack.data();
    float* cufft_begin = reinterpret_cast<float*>(cufft_compliant.data());
    
    for(size_t z = 0;z<shape[row_major::in_z];++z)
      for(size_t y = 0;y<shape[row_major::in_y];++y){
	
	size_t cufft_line_offset = (z*shape_for_cufft[row_major::in_y]*shape_for_cufft[row_major::in_x])+ (y*shape_for_cufft[row_major::in_x]);
	cufft_begin = reinterpret_cast<float*>(&cufft_compliant.data()[cufft_line_offset]);
	
	size_t stack_line_offset = (z*shape[row_major::in_y]*shape[row_major::in_x])+ (y*shape[row_major::in_x]);
	stack_begin = &_stack.data()[stack_line_offset];
	
	std::copy(stack_begin,stack_begin + shape[row_major::in_x],cufft_begin);
	
      }
    
    HANDLE_ERROR( hipMemcpy( d_stack, cufft_compliant.data(), size_for_cufft*sizeof(hipfftComplex) , hipMemcpyHostToDevice ) );

    //FORWARD
    hipfftHandle fftPlanFwd;
    hipfftPlan3d(&fftPlanFwd, shape[row_major::x], shape[row_major::y], shape[row_major::z], HIPFFT_R2C);HANDLE_ERROR_KERNEL;
    if(CUDART_VERSION < 6050)
      cufftSetCompatibilityMode(fftPlanFwd,CUFFT_COMPATIBILITY_FFTW_PADDING);

    hipfftExecR2C(fftPlanFwd, (hipfftReal*)d_stack, (hipfftComplex *)d_stack);HANDLE_ERROR_KERNEL;
    ( hipfftDestroy(fftPlanFwd) );HANDLE_ERROR_KERNEL;

    //apply scale
    const float scale_ = 1.f/float(img_size);
    unsigned threads = 32;
    unsigned blocks = (size_for_cufft + threads -1) /threads;
    scale<<<blocks,threads>>>(d_stack,size_for_cufft,scale_);
  
    //BACKWARD
    hipfftHandle fftPlanInv;
    hipfftPlan3d(&fftPlanInv, shape[row_major::x], shape[row_major::y], shape[row_major::z], HIPFFT_C2R);HANDLE_ERROR_KERNEL;
    if(CUDART_VERSION < 6050)
      cufftSetCompatibilityMode(fftPlanInv,CUFFT_COMPATIBILITY_FFTW_PADDING);
    
    hipfftExecC2R(fftPlanInv, (hipfftComplex*)d_stack, (hipfftReal *)d_stack);HANDLE_ERROR_KERNEL;
    ( hipfftDestroy(fftPlanInv) );HANDLE_ERROR_KERNEL;

    hipfftComplex zero;zero.x = 0;zero.y = 0;
    std::fill(cufft_compliant.data(),cufft_compliant.data()+cufft_compliant.num_elements(),zero);
    HANDLE_ERROR( hipMemcpy( cufft_compliant.data(), d_stack , size_for_cufft*sizeof(hipfftComplex) , hipMemcpyDeviceToHost ) );
    HANDLE_ERROR( hipFree( d_stack));

    for(size_t z = 0;z<shape[row_major::in_z];++z)
      for(size_t y = 0;y<shape[row_major::in_y];++y){
	
	size_t cufft_line_offset = (z*shape_for_cufft[row_major::in_y]*shape_for_cufft[row_major::in_x])+ (y*shape_for_cufft[row_major::in_x]);
	cufft_begin = reinterpret_cast<float*>(&cufft_compliant.data()[cufft_line_offset]);
	
	size_t stack_line_offset = (z*shape[row_major::in_y]*shape[row_major::in_x])+ (y*shape[row_major::in_x]);
	stack_begin = &_stack.data()[stack_line_offset];
	
	std::copy(cufft_begin,cufft_begin + shape[row_major::in_x],stack_begin);
	
      }

    return;
  }
  
  void outofplace_fft_ifft(const image_stack& _input,  image_stack& _output){

    
    std::vector<size_t> shape(_input.shape(),_input.shape() + 3);
    const size_t stack_size = _input.num_elements();

    if(_output.num_elements()!=stack_size)
      _output.resize(shape);
    
    std::fill(_output.data(),_output.data()+stack_size,0);
    
    std::vector<size_t> shape_for_cufft(shape);
    shape_for_cufft[row_major::x] = (shape[row_major::x]/2) + 1;
    size_t size_for_cufft = std::accumulate(shape_for_cufft.begin(), shape_for_cufft.end(),1,std::multiplies<size_t>());
  
    hipfftComplex* d_complex = 0;
    hipfftReal* d_real = 0;
  
    HANDLE_ERROR( hipMalloc( (void**)&(d_complex), size_for_cufft*sizeof(hipfftComplex) ) );
    HANDLE_ERROR( hipMemset( d_complex, 0, size_for_cufft*sizeof(hipfftComplex) ));

    HANDLE_ERROR( hipMalloc( (void**)&(d_real), stack_size*sizeof(hipfftComplex) ) );
    HANDLE_ERROR( hipMemcpy( d_real, _input.data(), stack_size*sizeof(float) , hipMemcpyHostToDevice ) );

    //FORWARD
    hipfftHandle fftPlanFwd;
    hipfftPlan3d(&fftPlanFwd, shape[row_major::x], shape[row_major::y], shape[row_major::z], HIPFFT_R2C);HANDLE_ERROR_KERNEL;
    if(CUDART_VERSION < 6050)
      cufftSetCompatibilityMode(fftPlanFwd,CUFFT_COMPATIBILITY_FFTW_PADDING);
    hipfftExecR2C(fftPlanFwd, d_real, d_complex);HANDLE_ERROR_KERNEL;

    //apply scale
    const float scale_ = 1.f/float(stack_size);
    unsigned threads = 32;
    unsigned blocks = (size_for_cufft + threads -1) /threads;
    scale<<<blocks,threads>>>(d_complex,size_for_cufft,scale_);
  
    //BACKWARD
    hipfftHandle fftPlanInv;
    hipfftPlan3d(&fftPlanInv, shape[row_major::x], shape[row_major::y], shape[row_major::z], HIPFFT_C2R);HANDLE_ERROR_KERNEL;
    if(CUDART_VERSION < 6050)
      cufftSetCompatibilityMode(fftPlanInv,CUFFT_COMPATIBILITY_FFTW_PADDING);

    hipfftExecC2R(fftPlanInv, d_complex, d_real);HANDLE_ERROR_KERNEL;
  
    std::fill(_output.data(),_output.data()+stack_size,0);
    HANDLE_ERROR( hipMemcpy( _output.data(), d_real , stack_size*sizeof(float) , hipMemcpyDeviceToHost ) );

    ( hipfftDestroy(fftPlanInv) );HANDLE_ERROR_KERNEL;
    ( hipfftDestroy(fftPlanFwd) );HANDLE_ERROR_KERNEL;

    HANDLE_ERROR( hipFree( d_real));
    HANDLE_ERROR( hipFree( d_complex));
  
  }
  
};

namespace fc = fourierconvolution;



BOOST_AUTO_TEST_SUITE(inplace)

BOOST_AUTO_TEST_CASE(of_prime_shape) {

  std::vector<size_t> shape(3,17);
  shape[fc::row_major::z] = 13;
  shape[fc::row_major::x] = 19;
  
  fc::image_stack stack(shape);

  for(size_t i = 0;i<stack.num_elements();++i)
    stack.data()[i] = i;

  fc::image_stack received(stack);

  fc::inplace_fft_ifft(received);
  
  double my_l2norm = l2norm(stack,received);
  const double expected = 1e-1;
  const bool result = my_l2norm<expected;

  if(!result && FC_TRACE){
    std::cout << boost::unit_test::framework::current_test_case().p_name << "\n";
    std::cout << "expected:\n";
    fc::print_stack(stack);
    std::cout << "\n\nreceived:\n";
    fc::print_stack(received);
  }
  
  BOOST_TEST_MESSAGE("inplace    shape(x,y,z)=" << shape[fc::row_major::x]<< ", " << shape[fc::row_major::y]<< ", " << shape[fc::row_major::z] << "\tl2norm = " << my_l2norm);
  BOOST_REQUIRE_MESSAGE(result,"l2norm = "<< my_l2norm <<" not smaller than " << expected);
  
  
}

BOOST_AUTO_TEST_CASE(power_of_2) {

  std::vector<size_t> shape(3,16);
  fc::image_stack stack(shape);

  for(size_t i = 0;i<stack.num_elements();++i)
    stack.data()[i] = i;

  fc::image_stack received(stack);

  fc::inplace_fft_ifft(received);
  
double my_l2norm = l2norm(stack,received);
  const double expected = 1e-1;
  const bool result = my_l2norm<expected;

  if(!result && FC_TRACE){
    std::cout << boost::unit_test::framework::current_test_case().p_name << "\n";
    std::cout << "expected:\n";
    fc::print_stack(stack);
    std::cout << "\n\nreceived:\n";
    fc::print_stack(received);
  }
  
  BOOST_TEST_MESSAGE("inplace    shape(x,y,z)=" << shape[fc::row_major::x]<< ", " << shape[fc::row_major::y]<< ", " << shape[fc::row_major::z] << "\tl2norm = " << my_l2norm);
  BOOST_REQUIRE_MESSAGE(result,"l2norm = "<< my_l2norm <<" not smaller than " << expected);
  
  
}


BOOST_AUTO_TEST_CASE(power_of_3) {

  std::vector<size_t> shape(3,27);
  fc::image_stack stack(shape);

  for(size_t i = 0;i<stack.num_elements();++i)
    stack.data()[i] = i;

  fc::image_stack received(stack);

  fc::inplace_fft_ifft(received);
  
double my_l2norm = l2norm(stack,received);
  const double expected = 1e-1;
  const bool result = my_l2norm<expected;

  if(!result && FC_TRACE){
    std::cout << boost::unit_test::framework::current_test_case().p_name << "\n";
    std::cout << "expected:\n";
    fc::print_stack(stack);
    std::cout << "\n\nreceived:\n";
    fc::print_stack(received);
  }
  
  BOOST_TEST_MESSAGE("inplace    shape(x,y,z)=" << shape[fc::row_major::x]<< ", " << shape[fc::row_major::y]<< ", " << shape[fc::row_major::z] << "\tl2norm = " << my_l2norm);
  BOOST_REQUIRE_MESSAGE(result,"l2norm = "<< my_l2norm <<" not smaller than " << expected);
  
  
}

BOOST_AUTO_TEST_CASE(power_of_5) {

  std::vector<size_t> shape(3,25);
  fc::image_stack stack(shape);

  for(size_t i = 0;i<stack.num_elements();++i)
    stack.data()[i] = i;

  fc::image_stack received(stack);

  fc::inplace_fft_ifft(received);
  
double my_l2norm = l2norm(stack,received);
  const double expected = 1e-1;
  const bool result = my_l2norm<expected;

  if(!result && FC_TRACE){
    std::cout << boost::unit_test::framework::current_test_case().p_name << "\n";
    std::cout << "expected:\n";
    fc::print_stack(stack);
    std::cout << "\n\nreceived:\n";
    fc::print_stack(received);
  }
  
  BOOST_TEST_MESSAGE("inplace    shape(x,y,z)=" << shape[fc::row_major::x]<< ", " << shape[fc::row_major::y]<< ", " << shape[fc::row_major::z] << "\tl2norm = " << my_l2norm);
  BOOST_REQUIRE_MESSAGE(result,"l2norm = "<< my_l2norm <<" not smaller than " << expected);
  
  
}

BOOST_AUTO_TEST_CASE(power_of_7) {

  std::vector<size_t> shape(3,2*7);
  fc::image_stack stack(shape);

  for(size_t i = 0;i<stack.num_elements();++i)
    stack.data()[i] = i;

  fc::image_stack received(stack);

  fc::inplace_fft_ifft(received);
  
double my_l2norm = l2norm(stack,received);
  const double expected = 1e-1;
  const bool result = my_l2norm<expected;

  if(!result && FC_TRACE){
    std::cout << boost::unit_test::framework::current_test_case().p_name << "\n";
    std::cout << "expected:\n";
    fc::print_stack(stack);
    std::cout << "\n\nreceived:\n";
    fc::print_stack(received);
  }
  
  BOOST_TEST_MESSAGE("inplace    shape(x,y,z)=" << shape[fc::row_major::x]<< ", " << shape[fc::row_major::y]<< ", " << shape[fc::row_major::z] << "\tl2norm = " << my_l2norm);
  BOOST_REQUIRE_MESSAGE(result,"l2norm = "<< my_l2norm <<" not smaller than " << expected);
  
  
}

BOOST_AUTO_TEST_CASE(cube_128_shape) {

  std::vector<size_t> shape(3,128);
  fc::image_stack stack(shape);

  for(size_t i = 0;i<stack.num_elements();++i)
    stack.data()[i] = i;

  fc::image_stack received(stack);

  fc::inplace_fft_ifft(received);
  
  double my_l2norm = l2norm(stack,received);
  const double expected = 1e-4;
  const bool result = my_l2norm<expected;

  if(!result && FC_TRACE){
    std::cout << boost::unit_test::framework::current_test_case().p_name << "\n";
    std::cout << "expected:\n";
    fc::print_stack(stack);
    std::cout << "\n\nreceived:\n";
    fc::print_stack(received);
  }
  
  BOOST_TEST_MESSAGE("inplace    shape(x,y,z)=" << shape[fc::row_major::x]<< ", " << shape[fc::row_major::y]<< ", " << shape[fc::row_major::z] << "\tl2norm = " << my_l2norm);
  BOOST_REQUIRE_MESSAGE(result,"l2norm = "<< my_l2norm <<" not smaller than " << expected);
  
  
}

BOOST_AUTO_TEST_SUITE_END()

BOOST_AUTO_TEST_SUITE(outofplace)

BOOST_AUTO_TEST_CASE(of_prime_shape) {

  std::vector<size_t> shape(3,17);
  shape[fc::row_major::z] = 13;
  shape[fc::row_major::x] = 19;

  fc::image_stack stack(shape);
  fc::image_stack received(shape);

  for(size_t i = 0;i<stack.num_elements();++i)
    stack.data()[i] = i;

  size_t img_size = std::accumulate(shape.begin(), shape.end(),1,std::multiplies<size_t>());

  BOOST_REQUIRE(img_size > 32);

  fc::outofplace_fft_ifft(stack, received);
  
double my_l2norm = l2norm(stack,received);

  const double expected = 1e-1;
  const bool result = my_l2norm<expected;
  if(!result && FC_TRACE){
    std::cout << "expected:\n";
    fc::print_stack(stack);
    std::cout << "\n\nreceived:\n";
    fc::print_stack(received);
  }
  
  BOOST_TEST_MESSAGE("outofplace shape(x,y,z)=" << shape[fc::row_major::x]<< ", " << shape[fc::row_major::y]<< ", " << shape[fc::row_major::z] << "\tl2norm = " << my_l2norm);
  BOOST_REQUIRE_MESSAGE(result,"l2norm = "<< my_l2norm <<" not smaller than " << expected);

}

BOOST_AUTO_TEST_CASE(power_of_2_shape) {

  std::vector<size_t> shape(3,16);

  fc::image_stack stack(shape);
  fc::image_stack received(shape);

  for(size_t i = 0;i<stack.num_elements();++i)
    stack.data()[i] = i;

  size_t img_size = std::accumulate(shape.begin(), shape.end(),1,std::multiplies<size_t>());

  BOOST_REQUIRE(img_size > 32);
  
  fc::outofplace_fft_ifft(stack, received);
double my_l2norm = l2norm(stack,received);
  const double expected = 1e-4;
  const bool result = my_l2norm<expected;
  
  if(!result && FC_TRACE){
    std::cout << "expected:\n";
    fc::print_stack(stack);
    std::cout << "\n\nreceived:\n";
    fc::print_stack(received);
    std::cout << "\nl2norm = " << my_l2norm << "\n";
  }

  BOOST_TEST_MESSAGE("outofplace shape(x,y,z)=" << shape[fc::row_major::x]<< ", " << shape[fc::row_major::y]<< ", " << shape[fc::row_major::z] << "\tl2norm = " << my_l2norm);
  BOOST_REQUIRE_MESSAGE(result,"l2norm = "<< my_l2norm <<" not smaller than " << expected);

}


BOOST_AUTO_TEST_CASE(power_of_3_shape) {

  std::vector<size_t> shape(3,std::pow(3,3));

  fc::image_stack stack(shape);
  fc::image_stack received(shape);

  for(size_t i = 0;i<stack.num_elements();++i)
    stack.data()[i] = i;

  size_t img_size = std::accumulate(shape.begin(), shape.end(),1,std::multiplies<size_t>());

  BOOST_REQUIRE(img_size > 32);
  
  fc::outofplace_fft_ifft(stack, received);
double my_l2norm = l2norm(stack,received);  const double expected = 1e-4;
  const bool result = my_l2norm<expected;
  if(!result && FC_TRACE){
    std::cout << "expected:\n";
    fc::print_stack(stack);
    std::cout << "\n\nreceived:\n";
    fc::print_stack(received);
    std::cout << "\nl2norm = " << my_l2norm << "\n";
  }

  BOOST_TEST_MESSAGE("outofplace shape(x,y,z)=" << shape[fc::row_major::x]<< ", " << shape[fc::row_major::y]<< ", " << shape[fc::row_major::z] << "\tl2norm = " << my_l2norm);
  BOOST_REQUIRE_MESSAGE(result,"l2norm = "<< my_l2norm <<" not smaller than " << expected);

}

BOOST_AUTO_TEST_CASE(power_of_5_shape) {

  std::vector<size_t> shape(3,std::pow(5,2));

  fc::image_stack stack(shape);
  fc::image_stack received(shape);

  for(size_t i = 0;i<stack.num_elements();++i)
    stack.data()[i] = i;

  size_t img_size = std::accumulate(shape.begin(), shape.end(),1,std::multiplies<size_t>());

  BOOST_REQUIRE(img_size > 32);
  
  fc::outofplace_fft_ifft(stack, received);
double my_l2norm = l2norm(stack,received);  const double expected = 1e-4;
  const bool result = my_l2norm<expected;
  if(!result && FC_TRACE){
    std::cout << "expected:\n";
    fc::print_stack(stack);
    std::cout << "\n\nreceived:\n";
    fc::print_stack(received);
    std::cout << "\nl2norm = " << my_l2norm << "\n";
  }

    BOOST_TEST_MESSAGE("outofplace shape(x,y,z)=" << shape[fc::row_major::x]<< ", " << shape[fc::row_major::y]<< ", " << shape[fc::row_major::z] << "\tl2norm = " << my_l2norm);
    BOOST_REQUIRE_MESSAGE(result,"l2norm = "<< my_l2norm <<" not smaller than " << expected);

}

BOOST_AUTO_TEST_CASE(power_of_7_shape) {

  std::vector<size_t> shape(3,14);

  fc::image_stack stack(shape);
  fc::image_stack received(shape);

  for(size_t i = 0;i<stack.num_elements();++i)
    stack.data()[i] = i;

  size_t img_size = std::accumulate(shape.begin(), shape.end(),1,std::multiplies<size_t>());

  BOOST_REQUIRE(img_size > 32);
  
  fc::outofplace_fft_ifft(stack, received);
double my_l2norm = l2norm(stack,received);  
  const double expected = 1e-3;
  const bool result = my_l2norm<expected;
  BOOST_TEST_MESSAGE("outofplace shape(x,y,z)=" << shape[fc::row_major::x]<< ", " << shape[fc::row_major::y]<< ", " << shape[fc::row_major::z] << "\tl2norm = " << my_l2norm);
  BOOST_REQUIRE_MESSAGE(result,"l2norm = "<< my_l2norm <<" not smaller than " << expected);

}

BOOST_AUTO_TEST_CASE(cube_128_shape) {

  std::vector<size_t> shape(3,128);

  fc::image_stack stack(shape);
  fc::image_stack received(shape);

  for(size_t i = 0;i<stack.num_elements();++i)
    stack.data()[i] = i;

  size_t img_size = std::accumulate(shape.begin(), shape.end(),1,std::multiplies<size_t>());

  BOOST_REQUIRE(img_size > 32);
  
  fc::outofplace_fft_ifft(stack, received);
  double my_l2norm = l2norm(stack,received);  
  const double expected = 1e-3;
  const bool result = my_l2norm<expected;
  BOOST_TEST_MESSAGE("outofplace shape(x,y,z)=" << shape[fc::row_major::x]<< ", " << shape[fc::row_major::y]<< ", " << shape[fc::row_major::z] << "\tl2norm = " << my_l2norm);
  BOOST_REQUIRE_MESSAGE(result,"l2norm = "<< my_l2norm <<" not smaller than " << expected);

}
BOOST_AUTO_TEST_SUITE_END()
