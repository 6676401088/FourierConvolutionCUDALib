/*

*/


// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <time.h>
#include <sys/time.h>
#include <hipfft/hipfft.h>
#include <iostream>
#include <fstream>

#include "../utils.h"

void cudasafe( hipError_t error, char* message = 0){
  if(error!=hipSuccess) { fprintf(stderr,"ERROR: %s : %i\n",message,error); exit(-1); }
}

void runTest(int argc, char **argv);

int main(int argc, char **argv)
{
    runTest(argc, argv);
}

void runTest(int argc, char **argv)
{

  hipSetDevice(0);

  long Nx = 256;
  long Ny = 256;
  long Nz = 256;
  
  if (argc==1){
	std::cout<<"usage: cufft_test Nx Ny Nz"<<std::endl;
    return;
  }
  
  if (argc>1)
    Nx = atol(argv[1]);
  
  if (argc>2)
    Ny  = atol(argv[2]);

  if (argc>3)
    Nz  = atol(argv[2]);

  const long Nz_half = Nz/2+1;

  printf("Nx = %ld Ny = %ld Nz = %ld \n",Nx,Ny, Nz);

  const long N_total = Nx*Ny*Nz;
  const long N_total_half = Nx*Ny*Nz_half;

  // the host buffer
  hipfftReal *x = (hipfftReal *)malloc(sizeof(hipfftReal) * N_total);
  hipfftReal *y = (hipfftReal *)malloc(sizeof(hipfftReal) * N_total);
  
  hipfftComplex *y_g;
  hipfftReal *x_g;

  cudasafe(hipMalloc((void **)&x_g,sizeof(hipfftReal) * N_total));
  cudasafe(hipMalloc((void **)&y_g,sizeof(hipfftComplex) * N_total_half));

  for (unsigned int i = 0; i < N_total; ++i)
    {
	  x[i] = 1.*rand()/RAND_MAX;
	}

	

  // Copy host memory to device
  cudasafe(hipMemcpy(x_g, x, sizeof(hipfftReal) * N_total, 
					  hipMemcpyHostToDevice));
	
  // CUFFT plan
  hipfftHandle plan_fwd, plan_bwd;
  hipfftPlan3d(&plan_fwd, Nx,Ny, Nz, HIPFFT_R2C);

  hipfftPlan3d(&plan_bwd, Nx,Ny, Nz, HIPFFT_C2R);

  hipfftExecR2C(plan_fwd, x_g, y_g);

  hipfftExecC2R(plan_bwd, y_g, x_g);


  hipMemcpy(y,x_g, sizeof(hipfftReal) * N_total, 
			 hipMemcpyDeviceToHost);

  //calc difference

  double diff = 0.;

  for (unsigned int i = 0; i < N_total; ++i)
	{
	  y[i] *= 1./N_total;
	  
	  diff += (x[i]-y[i])*(x[i]-y[i]);
	}

  printf("\nfirst element x: \t %f\n",x[0]);
  printf("\nfirst element y: \t %f\n",y[0]);

  printf("\nL2 difference: \t %f\n",diff);
  
  //Destroy CUFFT context
  hipfftDestroy(plan_fwd);
  hipfftDestroy(plan_bwd);

  // cleanup memory
  free(x);
  free(y);
  
  hipFree(x_g);
    	
  hipDeviceReset();
  exit(0);
}
