
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <iostream>
#include <fstream>

//#include "../utils.h"

inline static void cudasafe( hipError_t error, char* message = 0){
  if(error!=hipSuccess) { std::cerr << "ERROR: "<< message << " : " << error << "\n"; exit(1); }
}

void runTest(int argc, char **argv)
{

  hipSetDevice(0);

  long Nx = 256;
  long Ny = 256;
  long Nz = 256;
  
  if (argc==1){
    std::cout<<"usage: cufft_test Nx Ny Nz"<<std::endl;
    return;
  }
  
  if (argc>1)
    Nx = atol(argv[1]);
  
  if (argc>2)
    Ny  = atol(argv[2]);

  if (argc>3)
    Nz  = atol(argv[2]);


  printf("Nx = %ld Ny = %ld Nz = %ld \n",Nx,Ny, Nz);

  const long N_total = Nx*Ny*Nz;
  // the host buffer
  hipfftComplex *x = (hipfftComplex *)malloc(sizeof(hipfftComplex) * N_total);
  hipfftComplex *y = (hipfftComplex *)malloc(sizeof(hipfftComplex) * N_total);
  
  hipfftComplex *x_g;

  cudasafe(hipMalloc((void **)&x_g,sizeof(hipfftComplex) * N_total ));

  for (unsigned int i = 0; i < N_total; ++i)
    {
      x[i].x = 1.*rand()/RAND_MAX;
      x[i].y = 1.*rand()/RAND_MAX;

    }

	

  // Copy host memory to device
  cudasafe(hipMemcpy(x_g, x, sizeof(hipfftComplex) * N_total, 
		      hipMemcpyHostToDevice));
	
  // CUFFT plan
  hipfftHandle plan;
  hipfftPlan3d(&plan, Nx,Ny, Nz, HIPFFT_C2C);

  hipfftExecC2C(plan, x_g, x_g, HIPFFT_FORWARD);

  hipfftExecC2C(plan, x_g, x_g, HIPFFT_BACKWARD);


  hipMemcpy(y,x_g, sizeof(hipfftComplex) * N_total, 
	     hipMemcpyDeviceToHost);

  //calc difference

  double diff = 0.;

  for (unsigned int i = 0; i < N_total; ++i)
    {
      y[i].x *= 1./N_total;
      y[i].y *= 1./N_total;
	  
      diff += (x[i].x-y[i].x)*(x[i].x-y[i].x)+(x[i].y-y[i].y)*(x[i].y-y[i].y);
    }

  printf("\nfirst element x: \t %f + %fj\n",x[0].x,x[0].y);
  printf("\nfirst element y: \t %f + %fj\n",y[0].x,y[0].y);

  printf("\nL2 difference: \t %f\n",diff);
  
  //Destroy CUFFT context
  hipfftDestroy(plan);

  // cleanup memory
  free(x);
  free(y);
  
  hipFree(x_g);
    	
  hipDeviceReset();
  exit(0);
}
